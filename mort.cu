#include "hip/hip_runtime.h"
// Mort -- My Own RayTracer

#include <hip/hip_runtime.h>
#include <math.h>
#include <Windows.h>

#include "book.h"
#include "cpu_bitmap.h"
#include "gpu_anim.h"
#include "gl_helper.h"

#include "hittable_list.cuh"
#include "materials.cuh"
#include "objects.cuh"
#include "rng.cuh"
#include "vec3.cuh"
#include "camera.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define INF 2e10f
#define rnd(x) (x * rand() / RAND_MAX)

struct DataBlock {
	hipEvent_t start, stop;
	GPUAnimBitmap* bitmap;
	float totalTime;
	float frames;
	Camera cam;
	hiprandState* rand_states;
	int prevMouseX, prevMouseY;
	hittable_list data;
};

__global__ void renderKernel(Camera camera, uchar4* ptr, hiprandState* states, hittable_list world) {
	camera.render(ptr, states, world);
}

void input(DataBlock* d) {

	// Keyboard movement
	if (GetKeyState('W') & 0x8000) {
		d->cam.lookat += -(d->cam.w);
		d->cam.lookfrom += -(d->cam.w);
	}
	if (GetKeyState('S') & 0x8000) {
		d->cam.lookat += d->cam.w;
		d->cam.lookfrom += d->cam.w;
	}
	if (GetKeyState('A') & 0x8000) {
		d->cam.lookat += -(d->cam.u);
		d->cam.lookfrom += -(d->cam.u);
	}
	if (GetKeyState('D') & 0x8000) {
		d->cam.lookat += d->cam.u;
		d->cam.lookfrom += d->cam.u;
	}

	// Mouse movement
	POINT mousePos;
	GetCursorPos(&mousePos);
	int mouseDeltaX = mousePos.x - d->prevMouseX;
	int mouseDeltaY = mousePos.y - d->prevMouseY;

	if ((GetKeyState(VK_LBUTTON) & 0x8000) != 0) {
		if (mouseDeltaX != 0) {
			vec3 cam_direction = d->cam.lookat - d->cam.lookfrom;
			vec3 rotated = rotate_around(cam_direction, d->cam.vup, -mouseDeltaX / 500.0);
			d->cam.lookat = d->cam.lookfrom + rotated;
		}

		if (mouseDeltaY != 0) {
			vec3 cam_direction = d->cam.lookat - d->cam.lookfrom;
			vec3 rotated = rotate_around(cam_direction, d->cam.u, -mouseDeltaY / 500.0);
			d->cam.lookat = d->cam.lookfrom + rotated;
		}
	}
	d->prevMouseX = mousePos.x;
	d->prevMouseY = mousePos.y;
	d->cam.initialize();
}

void update(uchar4* output_bitmap, DataBlock* d, int ticks) {
	input(d);

	HANDLE_ERROR(hipEventRecord(d->start, 0));

	//// CUDA setup
	int width_blocks = ceil((float)d->cam.image_width / 16.0);
	int height_blocks = ceil((float)d->cam.image_height / 16.0);

	dim3 blocks(width_blocks, height_blocks);
	dim3 threads(16, 16);

	//// Render
	renderKernel <<<blocks, threads >>> (d->cam, output_bitmap, d->rand_states, d->data);
	//gpuErrchk(hipPeekAtLastError());
	//gpuErrchk(hipDeviceSynchronize());

	// FPS metrics
	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));

	d->totalTime += elapsedTime;
	++d->frames;

	//printf("Avg. time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock* d) {
	hipFree(d->rand_states);

	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

void random_spheres(hittable_list& data, Camera& cam) {
	solid_color checker1(color(.2, .3, .1));
	solid_color checker2(color(.9, .9, .9));
	checker_texture checker(0.32, checker1.getType(), checker1.getIdx(), checker2.getType(), checker2.getIdx());
	lambertian ground_material(checker.getType(), checker.getIdx());
	data.add(checker1);
	data.add(checker2);
	data.add(checker);
	data.add(ground_material);
	data.add(sphere(point3(0, -1000, 0), 1000, ground_material.getType(), ground_material.getIdx()));

	for (int a = -11; a < 11; a++) {
		for (int b = -11; b < 11; b++) {
			auto choose_mat = random_float();
			point3 center(a + 0.9 * random_float(), 0.2, b + 0.9 * random_float());

			if ((center - point3(4, 0.2, 0)).length() > 0.9) {
				if (choose_mat < 0.8) {
					// diffuse
					auto albedo = color::random() * color::random();
					auto center2 = center + vec3(0, random_float(0.0, 0.5), 0);
					solid_color color(albedo);
					lambertian material(color.getType(), color.getIdx());
					data.add(color);
					data.add(material);
					data.add(sphere(center, center2, 0.2, material.getType(), material.getIdx()));
				}
				else if (choose_mat < 0.95) {
					// metal
					auto albedo = color::random(0.5, 1);
					auto fuzz = random_float(0.0, 0.5);
					metal material(albedo, fuzz);
					data.add(material);
					data.add(sphere(center, 0.2, material.getType(), material.getIdx()));
				}
				else {
					// glass
					dielectric material(1.5);
					data.add(material);
					data.add(sphere(center, 0.2, material.getType(), material.getIdx()));
				}
			}
		}
	}

	dielectric material1(1.5);
	data.add(material1);
	data.add(sphere(point3(0, 1, 0), 1.0, material1.getType(), material1.getIdx()));

	solid_color sph_color(color(0.4, 0.2, 0.1));
	lambertian material2(sph_color.getType(), sph_color.getIdx());
	data.add(sph_color);
	data.add(material2);
	data.add(sphere(point3(-4, 1, 0), 1.0, material2.getType(), material2.getIdx()));

	metal material3(color(0.7, 0.6, 0.5), 0.0);
	data.add(material3);
	data.add(sphere(point3(4, 1, 0), 1.0, material3.getType(), material3.getIdx()));

	cam.aspect_ratio = 16.0 / 9.0;
	cam.image_width = 1200;
	cam.samples_per_pixel = 5;
	cam.bounce_limit = 5;

	cam.vfov = 20;
	cam.lookfrom = point3(13, 2, 3);
	cam.lookat = point3(0, 0, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0.0;
	cam.focus_dist = 10.0;

	return;
}

void two_spheres(hittable_list& data, Camera& cam) {
	solid_color checker1(color(.2, .3, .1));
	solid_color checker2(color(.9, .9, .9));
	checker_texture checker(0.32, checker1.getType(), checker1.getIdx(), checker2.getType(), checker2.getIdx());
	lambertian mat(checker.getType(), checker.getIdx());
	data.add(checker1);
	data.add(checker2);
	data.add(checker);
	data.add(mat);
	data.add(sphere(point3(0, -10, 0), 10, mat.getType(), mat.getIdx()));
	data.add(sphere(point3(0, 10, 0), 10, mat.getType(), mat.getIdx()));

	cam.aspect_ratio = 16.0 / 9.0;
	cam.image_width = 1200;
	cam.samples_per_pixel = 20;
	cam.bounce_limit = 50;

	cam.vfov = 20;
	cam.lookfrom = point3(13, 2, 3);
	cam.lookat = point3(0, 0, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

void earth(hittable_list& data, Camera& cam) {
	image_texture earth_texture("imgs/earthmap.jpg");
	lambertian earth_surface(earth_texture.getType(), earth_texture.getIdx());
	sphere globe(point3(0,0,0), 2, earth_surface.getType(), earth_surface.getIdx());
	data.add(earth_texture);
	data.add(earth_surface);
	data.add(globe);

	cam.aspect_ratio = 16.0 / 9.0;
	cam.image_width = 1200;
	cam.samples_per_pixel = 100;
	cam.bounce_limit = 50;

	cam.vfov = 20;
	cam.lookfrom = point3(0, 0, 12);
	cam.lookat = point3(0, 0, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

void two_perlin_spheres(hittable_list& data, Camera& cam) {
	noise_texture pertext(4.0);
	lambertian mat(pertext.getType(), pertext.getIdx());
	sphere s1(point3(0, -1000, 0), 1000, mat.getType(), mat.getIdx());
	sphere s2(point3(0, 2, 0), 2, mat.getType(), mat.getIdx());
	data.add(pertext);
	data.add(mat);
	data.add(s1);
	data.add(s2);

	cam.aspect_ratio = 16.0 / 9.0;
	cam.image_width = 1200;
	cam.samples_per_pixel = 5;
	cam.bounce_limit = 10;

	cam.vfov = 20;
	cam.lookfrom = point3(13, 2, 3);
	cam.lookat = point3(0, 0, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

void quads(hittable_list& data, Camera& cam) {
	solid_color red(color(1.0, 0.2, 0.2));
	solid_color green(color(0.2, 1.0, 0.2));
	solid_color blue(color(0.2, 0.2, 1.0));
	solid_color orange(color(1.0, 0.5, 0.0));
	solid_color teal(color(0.2, 0.8, 0.8));

	lambertian left_mat(red.getType(), red.getIdx());
	lambertian back_mat(green.getType(), green.getIdx());
	lambertian right_mat(blue.getType(), blue.getIdx());
	lambertian upper_mat(orange.getType(), orange.getIdx());
	lambertian lower_mat(teal.getType(), teal.getIdx());

	quad left_quad(point3(-3, -2, 5), vec3(0, 0, -4), vec3(0, 4, 0), left_mat.getType(), left_mat.getIdx());
	quad back_quad(point3(-2, -2, 0), vec3(4, 0, 0), vec3(0, 4, 0), back_mat.getType(), back_mat.getIdx());
	quad right_quad(point3(3, -2, 1), vec3(0, 0, 4), vec3(0, 4, 0), right_mat.getType(), right_mat.getIdx());
	quad upper_quad(point3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), upper_mat.getType(), upper_mat.getIdx());
	quad lower_quad(point3(-2, -3, 5), vec3(4, 0, 0), vec3(0, 0, -4), lower_mat.getType(), lower_mat.getIdx());

	data.add(red);
	data.add(green);
	data.add(blue);
	data.add(orange);
	data.add(teal);

	data.add(left_mat);
	data.add(back_mat);
	data.add(right_mat);
	data.add(upper_mat);
	data.add(lower_mat);

	data.add(left_quad);
	data.add(back_quad);
	data.add(right_quad);
	data.add(upper_quad);
	data.add(lower_quad);

	cam.aspect_ratio = 1.0;
	cam.image_width = 400;
	cam.samples_per_pixel = 100;
	cam.bounce_limit = 50;

	cam.vfov = 20;
	cam.lookfrom = point3(0, 0, 9);
	cam.lookat = point3(0, 0, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

void simple_light(hittable_list& data, Camera& cam) {

	noise_texture pertext(4);
	lambertian permat(pertext.getType(), pertext.getIdx());
	sphere s1(point3(0, -1000, 0), 1000, permat.getType(), permat.getIdx());
	sphere s2(point3(0, 2, 0), 2, permat.getType(), permat.getIdx());
	data.add(pertext);
	data.add(permat);
	data.add(s1);
	data.add(s2);

	solid_color lightcolor(color(4, 4, 4));
	diffuse_light difflight(lightcolor.getType(), lightcolor.getIdx());
	quad farQuad(point3(3, 1, -2), vec3(2, 0, 0), vec3(0, 2, 0), difflight.getType(), difflight.getIdx());
	sphere lightOrb(point3(0, 7, 0), 2, difflight.getType(), difflight.getIdx());
	data.add(lightcolor);
	data.add(difflight);
	data.add(farQuad);
	data.add(lightOrb);

	cam.aspect_ratio = 16.0 / 9.0;
	cam.image_width = 1200;
	cam.samples_per_pixel = 10;
	cam.bounce_limit = 10;
	cam.background = color(0.01, 0.01, 0.01);

	cam.vfov = 20;
	cam.lookfrom = point3(26, 3, 6);
	cam.lookat = point3(0, 2, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

void cornell_box(hittable_list& data, Camera& cam) {
	solid_color red(color(.65, .05, .05));
	solid_color white(color(.73, .73, .73));
	solid_color green(color(.12, .45, .15));
	solid_color light(color(15, 15, 10));

	lambertian red_wall(red.getType(), red.getIdx());
	lambertian white_wall(white.getType(), white.getIdx());
	lambertian green_wall(green.getType(), green.getIdx());
	diffuse_light lamp(light.getType(), light.getIdx());

	data.add(red);
	data.add(white);
	data.add(green);
	data.add(light);

	data.add(red_wall);
	data.add(white_wall);
	data.add(green_wall);
	data.add(lamp);

	data.add(quad(point3(555, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), green_wall.getType(), green_wall.getIdx()));
	data.add(quad(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), red_wall.getType(), red_wall.getIdx()));
	data.add(quad(point3(343, 554, 332), vec3(-130, 0, 0), vec3(0, 0, -105), lamp.getType(), lamp.getIdx()));
	data.add(quad(point3(0, 0, 0), vec3(555, 0, 0), vec3(0, 0, 555), white_wall.getType(), white_wall.getIdx()));
	data.add(quad(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), white_wall.getType(), white_wall.getIdx()));
	data.add(quad(point3(0, 0, 555), vec3(555, 0, 0), vec3(0, 555, 0), white_wall.getType(), white_wall.getIdx()));

	rotated_box(point3(130, 0, 65), point3(295, 165, 230), 15, white_wall.getType(), white_wall.getIdx(), data);
	rotated_box(point3(265, 0, 295), point3(430, 330, 460), -18, white_wall.getType(), white_wall.getIdx(), data);

	cam.aspect_ratio = 1.0;
	cam.image_width = 800;
	cam.samples_per_pixel = 1000;
	cam.bounce_limit = 500;
	cam.background = color(0, 0, 0);

	cam.vfov = 40;
	cam.lookfrom = point3(278, 278, -800);
	cam.lookat = point3(278, 278, 0);
	cam.vup = vec3(0, 1, 0);

	cam.defocus_angle = 0;
}

int main(void) {
	// Scene setup
	Camera cam;
	hittable_list data;

	int scene_idx = 7;

	switch(scene_idx) {
		case 1:
			random_spheres(data, cam);
			break;

		case 2:
			two_spheres(data, cam);
			break;

		case 3:
			earth(data, cam);
			break;

		case 4:
			two_perlin_spheres(data, cam);
			break;

		case 5:
			quads(data, cam);
			break;

		case 6:
			simple_light(data, cam);
			break;

		case 7:
			cornell_box(data, cam);
			break;
	}

	cam.initialize();
	data.toDevice();

	//// CUDA setup
	int width_blocks = ceil((float)cam.image_width / 16.0);
	int height_blocks = ceil((float)cam.image_height / 16.0);

	dim3 blocks(width_blocks, height_blocks);
	dim3 threads(16, 16);

	//// Change maximum CUDA stack size. Required to avoid an unspecified launch failure due to
	//// maximum stack size getting exceeded.
	HANDLE_ERROR(hipDeviceSetLimit(hipLimitStackSize, 2048));

	//// RNG initialisation
	hiprandState* dev_states;
	int seed = 69420;
	HANDLE_ERROR(hipMalloc((void**)&dev_states, cam.image_width * cam.image_height * sizeof(hiprandState)));
	setup_rng<<<blocks, threads>>>(dev_states, seed, cam.image_width);

	//// Recursion attenuation and emission setup
	color* recursionAttenuation;
	color* recursionEmission;
	HANDLE_ERROR(hipMalloc((void**)&recursionAttenuation, cam.bounce_limit * cam.image_width * cam.image_height * sizeof(color)));
	HANDLE_ERROR(hipMalloc((void**)&recursionEmission, cam.bounce_limit * cam.image_width * cam.image_height * sizeof(color)));
	cam.recursionAttenuation = recursionAttenuation;
	cam.recursionEmission = recursionEmission;

	//// Update function setup
	DataBlock update_data;
	GPUAnimBitmap bitmap(cam.image_width, cam.image_height, &update_data);
	update_data.bitmap = &bitmap;
	update_data.cam = cam;
	update_data.data = data;
	update_data.rand_states = dev_states;
	update_data.totalTime = 0;
	update_data.frames = 0;
	POINT mousePos;
	GetCursorPos(&mousePos);
	update_data.prevMouseX = mousePos.x;
	update_data.prevMouseY = mousePos.y;
	HANDLE_ERROR(hipEventCreate(&update_data.start));
	HANDLE_ERROR(hipEventCreate(&update_data.stop));

	bitmap.anim_and_exit((void (*)(uchar4*, void*, int))update, (void (*)(void*))anim_exit);
}